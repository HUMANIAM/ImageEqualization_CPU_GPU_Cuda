#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "hist-equ.h"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION

//time : 1266.4(ms)
//time : 7.45232(ms)
//time : 0.001920 (ms)

#include "stb-master\\stb_image.h"
#include "stb-master\\stb_image_write.h"
using namespace std;

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
const int  NUM_OF_THREADS = 1024; 

void run_cpu_color_test(PPM_IMG);
void run_gpu_color_test(PPM_IMG&, unsigned __int8 *, int , int);
void run_cpu_gray_test(PGM_IMG);
void run_gpu_gray_test(PGM_IMG);
PPM_IMG read_color_image(char*, unsigned __int8 * & );
void free_color_image(PPM_IMG&);
void wirte_color_image(char*, PPM_IMG&);
static void HandleError( hipError_t, const char *, int);
void calculate_lut(int * , unsigned __int8  * , int , int );
__global__ void histogram_equalization_GPU(unsigned __int8*, unsigned __int8*, int*, int);
__global__ void histogram_equalization__color__GPU(unsigned __int8* ,  unsigned __int8*, COLOR_LUT, int);
//************************************************************************

int main(){
 
	PGM_IMG img_ibuf_g;			   //input gray scale image
	PPM_IMG img_ibuf_c;			   //input color image
	unsigned __int8 * img_c_gpu;   //for gpu
	int n;

	//read gray scale image and color image
	img_ibuf_g.img = stbi_load("inputImages\\in-grayscale.jpg", &(img_ibuf_g.w), &(img_ibuf_g.h), &n, 1);
	img_ibuf_c = read_color_image("inputImages\\in-color.jpg", img_c_gpu);

	cout<<"Start Equalization process with grayscale image\n";
	run_cpu_gray_test(img_ibuf_g);
	run_gpu_gray_test(img_ibuf_g);

	cout<<"\n\nStart Equalization with color image\n";
	run_cpu_color_test(img_ibuf_c);
	run_gpu_color_test(img_ibuf_c, img_c_gpu, img_ibuf_c.w, img_ibuf_c.h);

	//free cpu memory 
	free(img_ibuf_g.img);
	free_color_image(img_ibuf_c);

    return 0;
}

void run_gpu_color_test(PPM_IMG& img_rgb,  unsigned __int8 * img_in, int w, int h)
{
	unsigned __int8 *  out_image;
	unsigned __int8 *  sharedImage;  // gpu can't access cpu memory so copy the image into shared memory
	COLOR_LUT lut;				    //hold lut for r g b axises
	float time;
	hipEvent_t start, stop;
	const int AR = w * h;
	const int SZ = AR * 3;
    
	hipMallocManaged(&(sharedImage), SZ * sizeof(unsigned __int8));
	hipMallocManaged(&(out_image), SZ * sizeof(unsigned __int8));
	//copy the color image in the shared memory
	for(int i=0; i<SZ; i++)
		sharedImage[i] = img_in[i];

	//lut for r axis
	hipMallocManaged(&(lut.lut_r), 256*sizeof(int));
	calculate_lut(lut.lut_r, img_rgb.img_r, AR, 256);
	//lut for g axis
	hipMallocManaged(&(lut.lut_g), 256*sizeof(int));
	calculate_lut(lut.lut_g, img_rgb.img_g, AR, 256);
	//lut for b axis
	hipMallocManaged(&(lut.lut_b), 256*sizeof(int));
	calculate_lut(lut.lut_b, img_rgb.img_b, AR, 256);

	//launch gpu threads to evaluate the image equalization
	const int NUM_OF_BLOCKS = (SZ+NUM_OF_THREADS-1)/NUM_OF_THREADS;
	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
	histogram_equalization__color__GPU<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(out_image, sharedImage, lut, SZ);
	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	cout<<"Processing time with GPU: "<<time<<"(ms)\n";
    
	stbi_write_png("outimages\\gpu_rgb.png", w, h, 3, out_image, 3*w);
	hipFree(lut.lut_r);
	hipFree(lut.lut_g);
	hipFree(lut.lut_b);
	hipFree(out_image);
}

void run_gpu_gray_test(PGM_IMG img_in)
{
    PGM_IMG result;
	float time;
	hipEvent_t start, stop;
	unsigned __int8 *sharedImage;  // gpu can't access cpu memory so copy the image into shared memory

	//evaluate the lut
	const int SZ = img_in.h * img_in.w;
	const int NUM_OF_BLOCKS = (SZ+NUM_OF_THREADS-1)/NUM_OF_THREADS;
	int *lut;
	hipMallocManaged(&lut, 256*sizeof(int));		//create lookup table for mapping from in to out
	calculate_lut(lut, img_in.img, SZ, 256);

	//equlize the image in the GPU
	result.w = img_in.w;
    result.h = img_in.h;
	hipMallocManaged(&(result.img), SZ * sizeof(unsigned __int8 ));
	hipMallocManaged(&sharedImage, SZ*sizeof(unsigned __int8));
 
	// copy from cpu memory to shared memroy
	for (int i = 0; i < SZ; i++)
		sharedImage[i] = img_in.img[i];

	//launch image equalization image in the GPU
	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
	
	histogram_equalization_GPU<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(result.img, sharedImage, lut, SZ);
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	cout<<"Processing time with GPU: "<<time<<"(ms)\n";
    
	stbi_write_png("outimages\\gpu_gray.png", result.w, result.h, 1, result.img, result.w);
	hipFree(result.img);
}

void run_cpu_color_test(PPM_IMG img_in)
{
    PPM_IMG img_obuf_hsl, img_obuf_yuv, img_obuf_rgb;
	float time;
	hipEvent_t start, stop;

	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
    img_obuf_rgb = contrast_enhancement_c_rgb(img_in);
    HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	cout<<"RGB Processing time with CPU: "<<time<<"(ms)\n";

	wirte_color_image("outimages\\cpu_rgb.png", img_obuf_rgb);
	free_color_image(img_obuf_rgb);
    
    HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
    img_obuf_hsl = contrast_enhancement_c_hsl(img_in);
    HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	cout<<"HSL Processing time with CPU: "<<time<<"(ms)\n";

	wirte_color_image("outimages\\hsv.png", img_obuf_hsl);
	free_color_image(img_obuf_hsl);

    HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
    img_obuf_yuv = contrast_enhancement_c_yuv(img_in);
    HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	cout<<"YUV Processing time with CPU: "<<time<<"(ms)\n";
    
   wirte_color_image("outimages\\yuv.png", img_obuf_yuv);
   free_color_image(img_obuf_yuv);
}


void run_cpu_gray_test(PGM_IMG img_in)
{
	PGM_IMG img_obuf;
	float time;
	hipEvent_t start, stop;
	
	HANDLE_ERROR( hipEventCreate(&start) );
	HANDLE_ERROR( hipEventCreate(&stop) );
	HANDLE_ERROR( hipEventRecord(start, 0) );
	img_obuf = contrast_enhancement_g(img_in);
	HANDLE_ERROR( hipEventRecord(stop, 0) );
	HANDLE_ERROR( hipEventSynchronize(stop) );
	HANDLE_ERROR( hipEventElapsedTime(&time, start, stop) );
	printf("Processing time with CPU: %f (ms)\n", time);
    
	stbi_write_png("outimages\\cpu_gray.png", img_obuf.w, img_obuf.h, 1, img_obuf.img, img_obuf.w);
	free(img_obuf.img);
}


PPM_IMG read_color_image(char* path, unsigned __int8 * & img){
	PPM_IMG img_ibuf_c;			//input color image
	int n, cimsz;
	//read color image
	img = stbi_load(path, &(img_ibuf_c.w), &(img_ibuf_c.h), &n, 3);

	cimsz = img_ibuf_c.w * img_ibuf_c.h * n;

	img_ibuf_c.img_r = (unsigned __int8 * ) malloc(img_ibuf_c.w * img_ibuf_c.h*sizeof(unsigned __int8));
	img_ibuf_c.img_g = (unsigned __int8 * ) malloc(img_ibuf_c.w * img_ibuf_c.h*sizeof(unsigned __int8));
	img_ibuf_c.img_b = (unsigned __int8 * ) malloc(img_ibuf_c.w * img_ibuf_c.h*sizeof(unsigned __int8));

	for(int i=0, j=0; i<cimsz; i+=3, j++){
		img_ibuf_c.img_r[j] = img[i];
		img_ibuf_c.img_g[j] = img[i+1];
		img_ibuf_c.img_b[j] = img[i+2];
	}
	return img_ibuf_c;
}

void free_color_image(PPM_IMG& im){
	free(im.img_b);
	free(im.img_g);
	free(im.img_r);
}

void wirte_color_image(char* path, PPM_IMG& img){
	int sz = img.w*img.h;
	unsigned __int8 * outimg = (unsigned __int8 *)malloc(3 * img.w * img.h * sizeof(unsigned __int8));

    for(int i = 0; i < sz; i ++){
        outimg[3*i + 0] = img.img_r[i];
        outimg[3*i + 1] = img.img_g[i];
        outimg[3*i + 2] = img.img_b[i];
    }

	stbi_write_png(path, img.w, img.h, 3, outimg, 3*img.w);
}

static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

void calculate_lut(int * lut, unsigned __int8  * img_in, int img_size, int nbr_bin){
	int i=0, min=0, d=0, cdf=0;

	//evaluate histogram
	int* hist = (int*) malloc(256*sizeof(int));		//commulative distributed funciton
	histogram(hist, img_in, img_size, nbr_bin);

    /* Construct the LUT by calculating the CDF */
	while(min == 0 && i<nbr_bin) min = hist[i++];
    d = img_size - min;

	//construct the lookup table
    for(i = 0; i < nbr_bin; i ++){
        cdf += hist[i];
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);

        if(lut[i] < 0)   lut[i] = 0;
		if(lut[i] > 255) lut[i] = 255;
    }

	//for(int i=0; i<nbr_bin; i++) cout<<i<<"   "<<lut[i]<<endl;
}

__global__ void histogram_equalization_GPU(unsigned __int8* img_out, unsigned __int8* img_in, int* lut, int img_size){
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	if(x<img_size)
		img_out[x] = lut[img_in[x]];
}

__global__ void histogram_equalization__color__GPU(unsigned __int8* img_out,  unsigned __int8* img_in, COLOR_LUT lut, int sz){
	int x = threadIdx.x + blockDim.x * blockIdx.x; 
	if(x<sz){
		if(x%3 == 0)
			img_out[x] = lut.lut_r[img_in[x]];
		else if(x%3 == 1)
			img_out[x] = lut.lut_g[img_in[x]];
		else 
			img_out[x] = lut.lut_b[img_in[x]];
	}
	
}